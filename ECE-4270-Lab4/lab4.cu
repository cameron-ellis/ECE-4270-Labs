
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float *d_out, float *d_a, float * d_b){
	int idx = threadIdx.x;
	float a = d_a[idx];
	float b = d_b[idx];
	d_out[idx] = 2*a + b*b;
}

int main(int argc, char ** argv){
	const int ARRAY_SIZE = 96;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
	
	float h_in[ARRAY_SIZE];
	for (int i=0; i < ARRAY_SIZE; i++){
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	float *d_in;
	float *d_in2;
	float *d_out;

	hipMalloc((void**) &d_in, ARRAY_BYTES); 
	hipMalloc((void**) &d_in2, ARRAY_BYTES); 
	hipMalloc((void**) &d_out, ARRAY_BYTES); 

	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_in2, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	square<<<1, ARRAY_SIZE>>>(d_out, d_in, d_in2);

	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	for (int i =0; i<ARRAY_SIZE; i++){
		printf("%f", h_out[i]);
		printf(((i % 4) !=3)? "\t": "\n");
	}
	
	hipFree(d_in);
	hipFree(d_in2);
	hipFree(d_out);

	return 0;
}